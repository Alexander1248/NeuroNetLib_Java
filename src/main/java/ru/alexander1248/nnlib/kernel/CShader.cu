
#include <hip/hip_runtime.h>
extern "C"

__global__ void calculate(float weightedSum, float *input, int *links, float *weights)
{
    int i = threadIdx.x;
    weightedSum += input[i] * links[i] * weights[i];
}