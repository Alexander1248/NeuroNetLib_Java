
#include <hip/hip_runtime.h>
extern "C"


__global__ void calculate(double weightedSum, double *input, double *weights, int *links)
{
    int i = threadIdx.x;
    weightedSum += input[i] * links[i] * weights[i];
}