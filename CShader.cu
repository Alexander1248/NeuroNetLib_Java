#include "hip/hip_runtime.h"


extern "C"
__global__ void calculate(int len, double weightedSum, double *input, double *weights, int *links)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) weightedSum += input[i] * links[i] * weights[i];
}